#include "hip/hip_runtime.h"
/*
Copyright (c) 2014-2015 Xiaowei Zhu, Tsinghua University

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

       http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/

#include "../core/graph_pr.cuh"
#include "../core/common.h"

#define GNUPLOT_PATH "/mnt/c/gnuplot/bin/gnuplot.exe"

int main(int argc, char ** argv) {
	if (argc<3) {
		fprintf(stderr, "usage: pagerank [path] [iterations] [memory budget in GB]\n");
		exit(-1);
	}
	std::string path = argv[1];
	int iterations = atoi(argv[2]);
	long memory_bytes = (argc>=4)?atol(argv[3])*1024l*1024l*1024l:8l*1024l*1024l*1024l;

	Graph graph(path);
	graph.set_memory_bytes(memory_bytes);
	BigVector<VertexId> degree(graph.path+"/degree", graph.vertices);
	BigVector<VertexId> in_edges(graph.path+"/in_edges", graph.vertices);
	BigVector<float> pagerank(graph.path+"/pagerank", graph.vertices);
	BigVector<float> sum(graph.path+"/sum", graph.vertices);

	long vertex_data_bytes = (long)graph.vertices * ( sizeof(VertexId) + sizeof(float) + sizeof(float) );
	graph.set_vertex_data_bytes(vertex_data_bytes);

	double begin_time = get_time();

	degree.fill(0);
	graph.stream_edges<VertexId>(
		[&](Edge & e){
			write_add(&degree[e.source], 1);
			return 0;
		}, nullptr, 0, 0
	);
	printf("degree calculation used %.2f seconds\n", get_time() - begin_time);

	in_edges.fill(0);
	graph.stream_edges<VertexId>(
		[&](Edge & e){
			write_add(&in_edges[e.target], 1);
			return 0;
		}, nullptr, 0, 0
	);
	int index;
	int max = 0;
	for (int i = 0; i < graph.vertices; i++) {
		if (max < in_edges[i]) {
			max = in_edges[i];
			index = i;
		}
	}
	printf("max=%d\n", max);
	printf("index=%d\n", index);

	BigVector<VertexId> in_edges_num(graph.path+"/in_edges_num", max+1);
	in_edges_num.fill(0);

	for (int i = 0; i < graph.vertices; i++) {
		in_edges_num[in_edges[i]]++;
	}

	FILE *gp;	// For gnuplot

	// gnuplotの起動コマンド
	if ((gp = popen(GNUPLOT_PATH, "w")) == NULL) {	// gnuplotをパイプで起動
		fprintf(stderr, "ファイルが見つかりません %s.", GNUPLOT_PATH);
		exit(EXIT_FAILURE);
	}

	// --- gnuplotにコマンドを送る --- //
	fprintf(gp, "set title \"Epinions\"\n");
	fprintf(gp, "set xrange [0:%d]\n", 4000); // 範囲の指定
	fprintf(gp, "set yrange [1:%d]\n", in_edges_num[0]);
	fprintf(gp, "set xlabel \"incoming edges\"\n"); // ラベル表示
	fprintf(gp, "set ylabel \"nodes\"\n");
	fprintf(gp, "unset key\n");
	// 点のプロット
	fprintf(gp, "plot '-' with points pointtype 6\n");
	for (int i = 0; i < max + 1; i++) {
		if (in_edges_num[i] != 0) fprintf(gp, "%d\t%d\n", i, in_edges_num[i]);
	}
	fprintf(gp, "e\n");
	fflush(gp); // バッファに格納されているデータを吐き出す（必須）
	// system("pause");
	for (int i = 0; i < 100; i++) {
		sleep(10);
	}
	fprintf(gp, "exit\n"); // gnuplotの終了
	pclose(gp);
	
	VertexId *degree_d;
	CHECK(hipMalloc((void**)&degree_d, sizeof(VertexId)*graph.vertices));
	CHECK(hipMemcpy(degree_d, degree.data, sizeof(VertexId)*graph.vertices, hipMemcpyHostToDevice));

	pagerank.fill(0);

	float *pagerank_d;
	CHECK(hipMalloc((void**)&pagerank_d, sizeof(float)*graph.vertices));
	CHECK(hipMemset(pagerank_d, 0, sizeof(float)*graph.vertices));

	float *sum_d;
	CHECK(hipMalloc((void**)&sum_d, sizeof(float)*graph.vertices));

	for (int iter=0; iter < iterations; iter++) {
		sum.fill(-INFINITY);
		CHECK(hipMemcpy(sum_d, sum.data, sizeof(float)*graph.vertices, hipMemcpyHostToDevice));
		graph.stream_edges_gpu<VertexId>(degree_d, pagerank_d, sum_d);
		// CHECK(hipMemcpy(sum.data, sum_d, sizeof(float)*graph.vertices, hipMemcpyDeviceToHost));
		// for (int i = 0; i < 10; i++) printf("sum[%d]=%f\n", i, sum[i]);
		graph.stream_vertices_gpu<VertexId>(pagerank_d, sum_d);
		// CHECK(hipMemcpy(pagerank.data, pagerank_d, sizeof(float)*graph.vertices, hipMemcpyDeviceToHost));
		// for (int i = 0; i < 10; i++) printf("pagerank[%d]=%f\n", i, pagerank[i]);
	}

	CHECK(hipMemcpy(pagerank.data, pagerank_d, sizeof(float)*graph.vertices, hipMemcpyDeviceToHost));
	for (int i = 0; i < 10; i++) printf("pagerank[%d]=%.2f\n", i, pagerank[i]);
}